#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "bubble/reduce.cuh"
#include "bubble/utils.cuh"
#include "dispatch_utils.h"
#include "kernel_version_utils.h"
#include "timer.cuh"

double reduce_add(torch::Tensor& out, torch::Tensor& input,
                  const std::string& version) {
  int hidden_size = input.size(0);

  TORCH_INTERNAL_ASSERT(
      version == "alpha" || version == "beta" || version == "delta",
      "The version is incorrect.");

  const at::cuda::OptionalCUDAGuard device_guard(input.device());
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  CUDAKernelTimer& timer = CUDAKernelTimer::getInstance();
  timer.begin();
  BUBBLE_DISPATCH_FLOATING_TYPES(input.scalar_type(), "reduce_add", [&] {
    int version_id = str2version(version);
    switch (version_id) {
      case 0: {
        int block_dim = std::min(hidden_size, 1024);
        int num_blocks = ceil_div(hidden_size, block_dim);
        int intermediate_size = pow(2, ceil(log2(block_dim)));
        torch::Tensor intermediate =
            torch::zeros({num_blocks, intermediate_size},
                         input.options().dtype(torch::kFloat32));
        bubble::alpha::reduce<scalar_t>(
            out.data_ptr<float>(), input.data_ptr<scalar_t>(),
            intermediate.data_ptr<float>(), hidden_size, stream);
      } break;
      case 1: {
        bubble::beta::reduce<scalar_t>(out.data_ptr<float>(),
                                       input.data_ptr<scalar_t>(),
                                       hidden_size, stream);
      } break;
      case 2: {
        bubble::delta::reduce<scalar_t>(out.data_ptr<float>(),
                                        input.data_ptr<scalar_t>(),
                                        hidden_size, stream);
      }
        break;
      default:
        std::cerr << "The version has not been supported yet." << std::endl;
        std::exit(-1);
    }
  });
  timer.end();
  return timer.elapse();
}

TORCH_LIBRARY_FRAGMENT(bubble, m) {
  m.def("bubble::reduce_add(Tensor! out, Tensor input, str version) -> float");
}

TORCH_LIBRARY_IMPL(bubble, CUDA, m) {
  m.impl("bubble::reduce_add", &reduce_add);
}
